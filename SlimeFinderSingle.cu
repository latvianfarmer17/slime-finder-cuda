
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <ctime>

#define cudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define SEED_RANGE(x) (x < -281474976710656 ? 0 : (x > 281474976710655 ? 0 : 1))

typedef long long int I64;
typedef long long unsigned U64;

typedef struct ChunkStruct {
    int x, z, isSlime;
} Chunk;

typedef struct DataStruct {
    int mode;
    I64 startSeed, endSeed;
    U64 totalSeeds;
    int rx, rz, rw, rh;
    int pw, ph, pl;
    Chunk pattern[32];
    int frequency;
} Data;

enum ErrorCode {
    errArgumentCount,
    errSeedNegativeRange,
    errSeedOutOfRange,
    errNotInteger,
    errInvalidMode,
    errPatternSize,
    errInvalidPattern,
    errInvalidFrequency,
    errNegativeInt,
    errCudaError,
    errUnexpectedError,
    errDLL,
    errFuncDLL
};

enum FinderMode {
    modePattern,
    modeFrequency,
    modeBenchmark
};

bool isStringInt(char *string);
I64 stringToInt(char *string);
bool stringEquate(char *a, char *b);
int stringLength(char *string);
bool parsePattern(Data *data, char *pattern);
bool parseFrequency(Data *data, char *frequency);
void exitError(int errorCode);

__device__ int isSlimeChunk(I64 seed, int x, int z);
__global__ void deviceTask(Data *data);

inline void __cudaSafeCall(hipError_t cError, const char *file, const int line);
void launchKernel(Data *data);

int main(int argc, char **argv) {
	Data data;

    if (argc == 9) {
        if (stringEquate(argv[1], "pattern")) {
            data.mode = modePattern;
        }
        else if (stringEquate(argv[1], "frequency")) {
            data.mode = modeFrequency;
        }
        else {
            exitError(errInvalidMode);
        }

        for (int i = 2; i < argc - 1; i++) {
            if (!isStringInt(argv[i])) {
                exitError(errNotInteger);
            }
        }

        data.startSeed = stringToInt(argv[2]);
        data.endSeed = stringToInt(argv[3]);
        data.rx = (int)stringToInt(argv[4]);
        data.rz = (int)stringToInt(argv[5]);
        data.rw = (int)stringToInt(argv[6]);
        data.rh = (int)stringToInt(argv[7]);

        if (data.startSeed > data.endSeed) exitError(errSeedNegativeRange);
        if (!SEED_RANGE(data.startSeed) || !SEED_RANGE(data.endSeed)) exitError(errSeedOutOfRange);

        if (data.mode == modeFrequency) {
            if (!parseFrequency(&data, argv[8])) exitError(errInvalidFrequency);
        }
        else {
            if (!parsePattern(&data, argv[8])) exitError(errInvalidPattern);
            if (data.rw < data.pw || data.rh < data.ph) exitError(errPatternSize);
        }
    }
    else if (argc == 2) {
        if (stringEquate(argv[1], "help")) {
            printf("SlimeFinder.exe help\n");
            printf("SlimeFinder.exe benchmark <total-seeds>\n");
            printf("SlimeFinder.exe pattern <start-seed> <end-seed> <rx> <rz> <rw> <rh> <pattern>\n");
            printf("SlimeFinder.exe frequency <start-seed> <end-seed> <rx> <rz> <rw> <rh> <frequency.srw.srh>\n");

            return 0;
        }
        else {
            exitError(errInvalidMode);
        }
    }
    else if (argc == 3) {
        if (stringEquate(argv[1], "benchmark")) {
            if (!isStringInt(argv[2])) exitError(errNotInteger);
            I64 totalSeeds = stringToInt(argv[2]);
            if (totalSeeds <= 0) exitError(errNegativeInt);

            srand(time(nullptr));

            data.mode = modeBenchmark;
            data.startSeed = rand();
            data.endSeed = data.startSeed + totalSeeds;
            data.totalSeeds = (U64)totalSeeds;
            data.rx = -250;
            data.rz = -250;
            data.rw = 500;
            data.rh = 500;
            data.pl = 0;

            int patternIndex = 0;

            for (int z = 0; z < 4; z++) {
                for (int x = 0; x < 4; x++) {
                    if ((x + z) % 2 == 0) {
                        data.pattern[patternIndex++] = { x, z, 1 };
                        data.pl++;
                    }
                }
            }
        }
        else {
            exitError(errInvalidMode);
        }
    }
    else {
        exitError(errArgumentCount);
    }

    launchKernel(&data);

    return 0;
}

bool isStringInt(char *string) {
    const char validChars[11] = "0123456789";
    int i = -1;

    while (string[++i] != '\0') {
        char c = string[i];
        bool valid = false;

        for (int j = 0; j < 11; j++) {
            if (c == validChars[j] || (i == 0 && c == '-')) {
                valid = true;
                break;
            }
        }

        if (!valid) return false;
    }

    return true;
}

I64 stringToInt(char *string) {
    const int negative = string[0] == '-';
    int length = stringLength(string);

    I64 n = 0;

    for (int i = negative ? 1 : 0; i < length; i++) {
        n += (I64)(pow(10, length - 1 - i) * (string[i] - '0'));
    }

    return negative ? -n : n;
}

bool stringEquate(char *a, char *b) {
    int aLen = stringLength(a);
    int bLen = stringLength(b);

    if (aLen != bLen) return false;

    for (int i = 0; i < aLen; i++) {
        if (a[i] != b[i]) return false;
    }

    return true;
}

int stringLength(char *string) {
    int length = -1;
    while (string[++length] != '\0');
    return length;
}

bool parsePattern(Data *data, char *pattern) {
    int patternLength = stringLength(pattern);
    int patternWidth = 0;
    int patternHeight = 0;
    int patternIndex = 0;

    for (int i = 0; i < patternLength; i++) {
        if (pattern[i] == '.') {
            break;
        }
        else {
            patternWidth++;
        }
    }

    char *token = strtok(pattern, ".");

    while (token != NULL) {
        if (stringLength(token) != patternWidth) return false;
        patternHeight++;
        token = strtok(NULL, ".");
    }

    data->pw = patternWidth;
    data->ph = patternHeight;
    data->pl = 0;

    int *patternLinear = (int *)malloc((patternWidth * patternHeight) * sizeof(int));
    
    if (patternLinear == NULL) return false;

    int patterLinearIndex = 0;

    for (int i = 0; i < patternLength; i++) {
        if (pattern[i] == '0' || pattern[i] == '1' || pattern[i] == '2') {
            patternLinear[patterLinearIndex++] = pattern[i] - '0';
        }
    }

    for (int z = 0; z < patternHeight; z++) {
        for (int x = 0; x < patternWidth; x++) {
            int patternValue = patternLinear[x + z * patternWidth];

            if (patternValue != 0 && patternValue != 1 && patternValue != 2) return false;
                   
            if (patternValue == 0 || patternValue == 1) {
                data->pattern[patternIndex++] = {x, z, patternValue};
                data->pl++;
            }
        }
    }

    free(patternLinear);

    return true;
}

bool parseFrequency(Data *data, char *frequency) {
    int f, pw, ph;
    int index = 0;

    char *token = strtok(frequency, ".");

    while (token != NULL) {
        switch (index) {
        case 0:
            if (!isStringInt(token)) return false;
            f = (int)stringToInt(token);
            if (f <= 0) return false;
            break;
        case 1:
            if (!isStringInt(token)) return false;
            pw = (int)stringToInt(token);
            if (pw <= 0) return false;
            break;
        case 2:
            if (!isStringInt(token)) return false;
            ph = (int)stringToInt(token);
            if (ph <= 0) return false;
            break;
        }

        index++;
        token = strtok(NULL, ".");
    }

    data->frequency = f;
    data->pw = pw;
    data->ph = ph;

    return f <= pw * ph;
}

void exitError(int errorCode) {
    switch (errorCode) {
    case errArgumentCount:
        printf("(%d) Error! Invalid argument count. Try using 'SlimeFinder.exe help'\n", errorCode);
        break;
    case errSeedNegativeRange:
        printf("(%d) Error! Start seed must be smaller than the end seed\n", errorCode);
        break;
    case errSeedOutOfRange:
        printf("(%d) Error! Seed must be between -281,474,976,710,656 and 281,474,976,710,655\n", errorCode);
        break;
    case errNotInteger:
        printf("(%d) Error! Argument must be an integer\n", errorCode);
        break;
    case errInvalidMode:
        printf("(%d) Error! Invalid mode selected. Try using 'SlimeFinder.exe help'\n", errorCode);
        break;
    case errInvalidPattern:
        printf("(%d) Error! Invalid pattern\n", errorCode);
        break;
    case errInvalidFrequency:
        printf("(%d) Error! Invalid frequency\n", errorCode);
        break;
    case errNegativeInt:
        printf("(%d) Error! Integer must be positive\n", errorCode);
        break;
    case errPatternSize:
        printf("(%d) Error! Pattern must fit in the region\n", errorCode);
        break;
    case errDLL:
        printf("(%d) Error! DLL could not be loaded\n", errorCode);
        break;
    case errFuncDLL:
        printf("(%d) Error! DLL function could not be loaded\n", errorCode);
        break;
    default:
        printf("(?) Error! Unexpected error\n");
        break;
    }

    exit(EXIT_FAILURE);
}

__device__ int isSlimeChunk(I64 seed, int x, int z) {
    seed += (I64)(x * x * 0x4c1906);
    seed += (I64)(x * 0x5ac0db);
    seed += (I64)(z * z) * 0x4307a7L;
    seed += (I64)(z * 0x5f24f);
    seed ^= 0x3ad8025fL;
    seed ^= 0x5deece66dL;
    seed &= 0xffffffffffff;
    return (((seed * 0x5deece66dL + 0xbL) & 0xffffffffffff) >> 17) % 10 == 0;
}

__global__ void deviceTask(Data *data) {
    int64_t seed = blockIdx.x * blockDim.x + threadIdx.x + data->startSeed;
    int x = blockIdx.y * blockDim.y + threadIdx.y + data->rx;
    int z = blockIdx.z * blockDim.z + threadIdx.z + data->rz;

    int validPattern, frequency;

    if (seed <= data->endSeed && x < data->rx + data->rw - 1 && z < data->rz + data->rh - 1) {
        switch (data->mode) {
        case modePattern:
            validPattern = 1;

            for (int i = 0; i < data->pl; i++) {
                Chunk p = data->pattern[i];
                if (isSlimeChunk(seed, x + p.x, z + p.z) != p.isSlime) {
                    validPattern = 0;
                    break;
                };
            }

            if (validPattern) {
                printf("        (+) Found seed -> %lld at (%d, %d) / (%d, %d)\a\n", seed, x, z, x << 4, z << 4);
            }

            break;
        case modeFrequency:
            frequency = 0;

            for (int px = 0; px < data->pw; px++) {
                for (int pz = 0; pz < data->ph; pz++) {
                    if (isSlimeChunk(seed, x + px, z + pz)) frequency++;
                }
            }

            if (frequency >= data->frequency) {
                printf("        (+) Found seed -> %lld at (%d, %d) / (%d, %d) with frequency %d\a\n", seed, x, z, x << 4, z << 4, frequency);
            }

            break;
        case modeBenchmark:
            validPattern = 1;

            for (int i = 0; i < data->pl; i++) {
                Chunk p = data->pattern[i];
                if (isSlimeChunk(seed, x + p.x, z + p.z) != p.isSlime) {
                    validPattern = 0;
                    break;
                }
            }

            break;
        }
    }
}

inline void __cudaSafeCall(hipError_t cError, const char *file, const int line) {
    if (cError != hipSuccess) {
        printf("(%d) Error! '%s' CUDA error %d\n", errCudaError, hipGetErrorName(cError), (int)cError);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

void launchKernel(Data *data) {
    int deviceId;
    hipDeviceProp_t prop;

    cudaSafeCall(hipGetDevice(&deviceId));
    cudaSafeCall(hipGetDeviceProperties(&prop, deviceId));

    const I64 startSeed = data->startSeed;
    const I64 endSeed = data->endSeed;
    const int xStart = data->rx;
    const int zStart = data->rz;
    const int xRange = data->rw;
    const int zRange = data->rh;

    int tpb = 1 << (int)log2(pow(prop.maxThreadsPerBlock, 0.3333333333333333f));

    dim3 threadsPerBlock(tpb, tpb, tpb);

    U64 numBlocksSeed = ((endSeed - startSeed) + threadsPerBlock.x - 1) / threadsPerBlock.x;
    U64 numBlocksX = (xRange + threadsPerBlock.y - 1) / threadsPerBlock.y;
    U64 numBlocksZ = (zRange + threadsPerBlock.z - 1) / threadsPerBlock.z;

    U64 seedMaxBlocks = prop.maxGridSize[0] >> 4;
    U64 seedRemainder = numBlocksSeed % seedMaxBlocks;
    int seedTotalChunks = (int)((numBlocksSeed - seedRemainder) / seedMaxBlocks);

    U64 seedRange = endSeed - startSeed;
    U64 seedRangeRemainder = seedRange % (seedMaxBlocks * tpb);
    U64 seedsPerChunk = (seedRange - seedRangeRemainder) / (seedTotalChunks + 1);

    U64 xMaxBlocks = prop.maxGridSize[1] >> 4;
    U64 xRemainder = numBlocksX % xMaxBlocks;
    int xTotalChunks = (int)((numBlocksX - xRemainder) / xMaxBlocks);

    U64 xRangeRemainder = xRange % (xMaxBlocks * tpb);
    U64 xPerChunk = xRange / (xTotalChunks + 1);

    U64 zMaxBlocks = prop.maxGridSize[2] >> 4;
    U64 zRemainder = numBlocksZ % zMaxBlocks;
    int zTotalChunks = (int)((numBlocksZ - zRemainder) / zMaxBlocks);

    U64 zRangeRemainder = zRange % (zMaxBlocks * tpb);
    U64 zPerChunk = zRange / (zTotalChunks + 1);

    Data *dataDevice;
    hipEvent_t startEvent, stopEvent;

    if (data->mode == modeBenchmark) {
        cudaSafeCall(hipEventCreate(&startEvent));
        cudaSafeCall(hipEventCreate(&stopEvent));
    }
    else {
        printf("(?) Device      | %s\n", prop.name);
        printf("(?) Mode        | %s\n", data->mode == modePattern ? "Pattern" : "Frequency");
        printf("(?) Seed range  | %lld to %lld\n", data->startSeed, data->endSeed);
        printf("(?) Total chunks| (%d, %d, %d)\n", seedTotalChunks + 1, xTotalChunks + 1, zTotalChunks + 1);
    }

    for (int sc = 0; sc <= seedTotalChunks; sc++) {
        for (int xc = 0; xc <= xTotalChunks; xc++) {
            for (int zc = 0; zc <= zTotalChunks; zc++) {
                if (data->mode != modeBenchmark) {
                    printf("    (!) Computing data chunk (%d, %d, %d)\n", sc, xc, zc);
                }
                else {
                    printf("(!) Benchmarking...\n");
                }

                dim3 numBlocks(sc == seedTotalChunks ? seedRemainder : seedMaxBlocks, xc == xTotalChunks ? xRemainder : xMaxBlocks, zc == zTotalChunks ? zRemainder : zMaxBlocks);

                data->startSeed = (sc * seedsPerChunk) + startSeed;
                data->endSeed = data->startSeed + seedsPerChunk + (sc == seedTotalChunks ? seedRangeRemainder : 0);

                data->rx = (xc * xPerChunk) + xStart;
                data->rz = (zc * zPerChunk) + zStart;
                data->rw = (xc * xPerChunk) + (xc == xTotalChunks ? xRangeRemainder : xPerChunk);
                data->rh = (zc * zPerChunk) + (zc == zTotalChunks ? zRangeRemainder : zPerChunk);

                cudaSafeCall(hipMalloc((void **)&dataDevice, sizeof(Data)));
                cudaSafeCall(hipMemcpy(dataDevice, data, sizeof(Data), hipMemcpyHostToDevice));

                if (data->mode == modeBenchmark) cudaSafeCall(hipEventRecord(startEvent));

                deviceTask <<< numBlocks, threadsPerBlock >>> (dataDevice);

                if (data->mode == modeBenchmark) {
                    cudaSafeCall(hipEventRecord(stopEvent));
                    cudaSafeCall(hipEventSynchronize(stopEvent));

                    float timeTaken = 0;

                    cudaSafeCall(hipEventElapsedTime(&timeTaken, startEvent, stopEvent));

                    uint64_t seedRate = (uint64_t)(data->totalSeeds * 247009000) / (uint64_t)timeTaken;

                    printf("(?) Benchmark took %f ms which is approximately %llu pattern checks per second\n", timeTaken, seedRate);
                    printf("    (?) The pattern has dimensions of 4x4 with 50%% of chunks being slime chunks\n");
                    printf("    (?) A region of (-250, -250, 500, 500) was checked with %llu seeds\n", data->totalSeeds);
                    printf("    (?) So a region of (-50, -50, 100, 100) could check %llu seeds instead\n", seedRate / 9409);
                    printf("        with the same pattern in the same amount of time (roughly speaking)\n");
                    printf("\n    Slime chunks | Avg. Time (min)\n");

                    for (int i = 1; i <= 20; i++) {
                        float t = pow(10, i) / ((float)seedRate * 60.0f);
                        if (t <= 30000.0f) {
                            printf("         %-7d |    %f\n", i, t);
                        }
                        else {
                            printf("         %-7d |    Long...\n", i);
                        }
                    }
                }
                else {
                    cudaSafeCall(hipDeviceSynchronize());
                }

                cudaSafeCall(hipFree(dataDevice));
            }
        }
    }

    hipDeviceReset();
}
