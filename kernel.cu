// NOTE: I tried to make header files to nicely structure the project but I just could not figure out how to link CUDA and C/C++ code...
//       So enjoy going through this if you ever need to (or want to)

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include <string.h>
#include <math.h>

#define SEED_RANGE(x) (x < -281474976710656 ? 0 : (x > 281474976710655 ? 0 : 1))
#define cudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)

typedef struct vec2PatternStruct {
    int x, z, type;
} vec2Pattern;

typedef struct sfDataStruct {
    int mode;
    int64_t startSeed, endSeed;
    int64_t worldSeed;
    int rx, rz, rw, rh;
    int pw, ph, pl;
    vec2Pattern pattern[32];
    int frequency;
} sfData;

enum ErrorCode {
    errArgumentCount,
    errSeedNegativeRange,
    errSeedOutOfRange,
    errNotInteger,
    errInvalidMode,
    errPatternSize,
    errInvalidPattern,
    errInvalidFrequency,
    errCudaError,
    errUnexpectedError
};

enum FinderMode {
    modePattern,
    modeFrequency
};

int isStringInt(char *string);
int64_t stringToInt(char *string);
int stringEquate(char *a, char *b);
int stringLength(char *string);
int parsePattern(sfData *data, char *pattern);
int parseFrequency(sfData *data, char *frequency);
inline void __cudaSafeCall(hipError_t err, const char *file, const int line);
void exitError(int errorCode);
void launchKernel(sfData *data);

__device__ int isSlimeChunk(int64_t seed, int x, int z);
__device__ int checkPattern(sfData *data, int64_t seed, int xOff, int zOff);
__device__ int checkFrequency(sfData *data, int64_t seed, int xOff, int zOff);
__global__ void deviceTask(sfData *data);

// Code which emulates Minecraft's slime chunk determination
__device__ int isSlimeChunk(int64_t seed, int x, int z) {
    seed += (int64_t)(x * x * 0x4c1906);
    seed += (int64_t)(x * 0x5ac0db);
    seed += (int64_t)(z * z) * 0x4307a7L;
    seed += (int64_t)(z * 0x5f24f);
    seed ^= 0x3ad8025fL;

    seed ^= 0x5deece66dL;
    seed &= 0xffffffffffff;

    int bits, val;

    do {
        seed = (seed * 0x5deece66dL + 0xbL) & 0xffffffffffff;
        bits = (int)(seed >> 17);
        val = bits % 10;
    } while (bits - val + 9 < 0);

    return val == 0;
}

// Checks if the given pattern is valid for the world seed and the current region co-ordinate offset
__device__ int checkPattern(sfData *data, int64_t seed, int xOff, int zOff) {
    for (int i = 0; i < data->pl; i++) {
        vec2Pattern p = data->pattern[i];
        if (isSlimeChunk(seed, xOff + p.x, zOff + p.z) != p.type) return 0;
    }

    return 1;
}

// Counts the total slime chunks within a subset of the region
__device__ int checkFrequency(sfData *data, int64_t seed, int xOff, int zOff) {
    int chunksFound = 0;

    for (int x = 0; x < data->pw; x++) {
        for (int z = 0; z < data->ph; z++) {
            if (isSlimeChunk(seed, xOff + x, zOff + z)) chunksFound++;
        }
    }

    return chunksFound;
}

// 'Main' kernel function, evaluates the slime finder mode and looks for slime chunks accordingly
__global__ void deviceTask(sfData *data) {
    int64_t seed = blockIdx.x * blockDim.x + threadIdx.x + data->startSeed;
    int x = blockIdx.y * blockDim.y + threadIdx.y + data->rx;
    int z = blockIdx.z * blockDim.z + threadIdx.z + data->rz;

    if (seed <= data->endSeed && x < data->rx + data->rw - 1 && z < data->rz + data->rh - 1) {
        switch (data->mode) {
        case modePattern:
            if (checkPattern(data, seed, x, z)) {
                printf("    (+) Found seed -> %lld at (%d, %d) / (%d, %d)\a\n", seed, x, z, x << 4, z << 4);
            }

            break;
        case modeFrequency:
            int frequency = checkFrequency(data, seed, x, z);

            if (frequency >= data->frequency) {
                printf("    (+) Found seed -> %lld at (%d, %d) / (%d, %d) with frequency %d\a\n", seed, x, z, x << 4, z << 4, frequency);
            }

            break;
        }
    }
}

int main(int argc, char **argv) {
    sfData data;

    if (argc == 9) {
        // Set the slime finder mode
        if (stringEquate(argv[1], "pattern")) {
            data.mode = modePattern;
        }
        else if (stringEquate(argv[1], "frequency")) {
            data.mode = modeFrequency;
        }
        else {
            exitError(errInvalidMode);
        }

        // Check all arguments are integers (except the last one and first two)
        for (int i = 2; i < argc - 1; i++) {
            if (!isStringInt(argv[i])) {
                exitError(errNotInteger);
            }
        }

        // Populate the slime finder data
        data.startSeed = stringToInt(argv[2]);
        data.endSeed = stringToInt(argv[3]);
        data.rx = (int)stringToInt(argv[4]);
        data.rz = (int)stringToInt(argv[5]);
        data.rw = (int)stringToInt(argv[6]);
        data.rh = (int)stringToInt(argv[7]);

        if (data.startSeed > data.endSeed) exitError(errSeedNegativeRange);
        if (!SEED_RANGE(data.startSeed) || !SEED_RANGE(data.endSeed)) exitError(errSeedOutOfRange);

        if (data.mode == modeFrequency) {
            if (!parseFrequency(&data, argv[8])) exitError(errInvalidFrequency);
        }
        else {
            if (!parsePattern(&data, argv[8])) exitError(errInvalidPattern);
            if (data.rw < data.pw || data.rh < data.ph) exitError(errPatternSize);
        }
    }
    else if (argc == 2) {
        const char helpString[5] = "help";
        int length = stringLength(argv[1]);

        if (length != 4) exitError(errInvalidMode);

        for (int i = 0; i < 4; i++) {
            if (helpString[i] != argv[1][i]) exitError(errInvalidMode);
        }

        printf("\nSlimeFinder.exe <mode=pattern>   <start-seed> <end-seed> <rx> <rz> <rw> <rh> <pattern>\n");
        printf("SlimeFinder.exe <mode=frequency> <start-seed> <end-seed> <rx> <rz> <rw> <rh> <frequency.srw.srh>\n\n");

        return 0;
    }
    else {
        exitError(errArgumentCount);
    }

    launchKernel(&data);

    hipDeviceReset();

    return 0;
}

// Check if a string is in a valid integer format
int isStringInt(char *string) {
    const char validChars[11] = "0123456789";
    int i = -1;

    while (string[++i] != '\0') {
        char c = string[i];
        int valid = 0;

        for (int j = 0; j < 11; j++) {
            if (c == validChars[j] || (i == 0 && c == '-')) {
                valid = 1;
                break;
            }
        }

        if (!valid) return 0;
    }

    return 1;
}

// Convert a string to an integer
int64_t stringToInt(char *string) {
    const int negative = string[0] == '-';
    int length = stringLength(string);

    int64_t n = 0;

    for (int i = negative ? 1 : 0; i < length; i++) {
        n += (int64_t)(pow(10, length - 1 - i) * (string[i] - '0'));
    }

    return negative ? -n : n;
}

// Compare if strings a and b are the same
int stringEquate(char *a, char *b) {
    int aLen = stringLength(a);
    int bLen = stringLength(b);

    if (aLen != bLen) return 0;

    for (int i = 0; i < aLen; i++) {
        if (a[i] != b[i]) return 0;
    }

    return 1;
}

// Compute the string length
int stringLength(char *string) {
    int length = -1;
    while (string[++length] != '\0');
    return length;
}

// Process the user pattern input
int parsePattern(sfData *data, char *pattern) {
    int patternLength = stringLength(pattern);
    int patternWidth = 0;
    int patternHeight = 0;
    int patternIndex = 0;

    for (int i = 0; i < patternLength; i++) {
        if (pattern[i] == '.') {
            break;
        }
        else {
            patternWidth++;
        }
    }

    char *token = strtok(pattern, ".");

    while (token != NULL) {
        if (stringLength(token) != patternWidth) return 0;
        patternHeight++;
        token = strtok(NULL, ".");
    }

    data->pw = patternWidth;
    data->ph = patternHeight;
    data->pl = 0;

    for (int z = 0; z < patternHeight; z++) {
        for (int x = 0; x < patternWidth; x++) {
            int patternValue = pattern[x + z * patternHeight + z] - '0';

            if (patternValue != 0 && patternValue != 1 && patternValue != 2) return 0;
            
            if (patternValue == 0 || patternValue == 1) {
                data->pattern[patternIndex++] = {x, z, patternValue};
                data->pl++;
            }
        }
    }

    return 1;
}

// Process the user frequency input
int parseFrequency(sfData *data, char *frequency) {
    int f, pw, ph;
    int index = 0;

    char *token = strtok(frequency, ".");

    while (token != NULL) {
        switch (index) {
        case 0:
            if (!isStringInt(token)) return 0;
            f = (int)stringToInt(token);
            break;
        case 1:
            if (!isStringInt(token)) return 0;
            pw = (int)stringToInt(token);
            break;
        case 2:
            if (!isStringInt(token)) return 0;
            ph = (int)stringToInt(token);
            break;
        }

        index++;
        token = strtok(NULL, ".");
    }

    data->frequency = f;
    data->pw = pw;
    data->ph = ph;

    return f <= pw * ph;
}

// Processes any CUDA function and evaluates the returned error (if any)
inline void __cudaSafeCall(hipError_t cError, const char *file, const int line) {
    if (cError != hipSuccess) {
        printf("(%d) Error! '%s' CUDA error %d\n", errCudaError, hipGetErrorName(cError), (int)cError);
        hipDeviceReset();
        exit(errCudaError);
    }
}

// Exit the program with a specific error message
void exitError(int errorCode) {
    switch (errorCode) {
    case errArgumentCount:
        printf("(%d) Error! Invalid argument count. Try using 'SlimeFinder.exe help'\n", errorCode);
        break;
    case errSeedNegativeRange:
        printf("(%d) Error! Start seed must be smaller than the end seed\n", errorCode);
        break;
    case errSeedOutOfRange:
        printf("(%d) Error! Seed must be between -281,474,976,710,656 and 281,474,976,710,655\n", errorCode);
        break;
    case errNotInteger:
        printf("(%d) Error! Argument must be an integer\n", errorCode);
        break;
    case errInvalidMode:
        printf("(%d) Error! Invalid mode selected. Try using 'SlimeFinder.exe help'\n", errorCode);
        break;
    case errInvalidPattern:
        printf("(%d) Error! Invalid pattern\n", errorCode);
        break;
    case errInvalidFrequency:
        printf("(%d) Error! Invalid frequency\n", errorCode);
        break;
    case errPatternSize:
        printf("(%d) Error! Pattern must fit in the region\n", errorCode);
        break;
    default:
        printf("(?) Error! Unexpected error\n");
        exit(errUnexpectedError);
    }

    hipDeviceReset();

    exit(errorCode);
}

// Launches the kernel
void launchKernel(sfData *data) {
    int deviceId;
    hipDeviceProp_t prop;
    
    cudaSafeCall(hipGetDevice(&deviceId));
    cudaSafeCall(hipGetDeviceProperties(&prop, deviceId));

    const int64_t startSeed = data->startSeed;
    const int64_t endSeed = data->endSeed;
    const int xStart = data->rx;
    const int zStart = data->rz;
    const int xRange = data->rw;
    const int zRange = data->rh;

    // Calculate the highest amount of threads per block possible
    int tpb = (int)pow(prop.maxThreadsPerBlock, 0.3333333333333333f);
    
    dim3 threadsPerBlock(tpb, tpb, tpb);

    // Calculate the number of blocks for each grid dimension
    uint64_t numBlocksSeed = ((endSeed - startSeed + 1) + threadsPerBlock.x - 1) / threadsPerBlock.x;
    uint64_t numBlocksX = (xRange + threadsPerBlock.y - 1) / threadsPerBlock.y;
    uint64_t numBlocksZ = (zRange + threadsPerBlock.z - 1) / threadsPerBlock.z;
    
    // Calculate the total amount of chunks and ranges
    //  Seed
    uint64_t seedMaxBlocks = prop.maxGridSize[0];
    uint64_t seedRemainder = numBlocksSeed % seedMaxBlocks;
    int seedTotalChunks = (int)((numBlocksSeed - seedRemainder) / seedMaxBlocks);
    
    uint64_t seedRange = endSeed - startSeed;
    uint64_t seedRangeRemainder = seedRange % (seedMaxBlocks * tpb);
    uint64_t seedsPerChunk = (seedRange - seedRangeRemainder) / (seedTotalChunks + 1);

    //  X
    uint64_t xMaxBlocks = prop.maxGridSize[1];
    uint64_t xRemainder = numBlocksX % xMaxBlocks;
    int xTotalChunks = (int)((numBlocksX - xRemainder) / xMaxBlocks);

    uint64_t xRangeRemainder = xRange % (xMaxBlocks * tpb);
    uint64_t xPerChunk = (xRange - xRangeRemainder) / (xMaxBlocks + 1);

    //  Z
    uint64_t zMaxBlocks = prop.maxGridSize[2];
    uint64_t zRemainder = numBlocksZ % zMaxBlocks;
    int zTotalChunks = (int)((numBlocksZ - zRemainder) / zMaxBlocks);

    uint64_t zRangeRemainder = zRange % (zMaxBlocks * tpb);
    uint64_t zPerChunk = (zRange - zRangeRemainder) / (zMaxBlocks + 1);

    // Distribute the task across "data chunks" if there is enough data to process
    sfData *dataDevice;

    for (int sc = 0; sc <= seedTotalChunks; sc++) {
        for (int xc = 0; xc <= xTotalChunks; xc++) {
            for (int zc = 0; zc <= zTotalChunks; zc++) {
                printf("(?) Computing data chunk %d|%d|%d\n", sc, xc, zc);

                dim3 numBlocks(sc == seedTotalChunks ? seedRemainder : seedMaxBlocks, xc == xTotalChunks ? xRemainder : xMaxBlocks, zc == zTotalChunks ? zRemainder : zMaxBlocks);
                
                data->startSeed = (sc * seedsPerChunk) + startSeed;
                data->endSeed = data->startSeed + seedsPerChunk + (sc == seedTotalChunks ? seedRangeRemainder : 0);

                data->rx = (xc * xPerChunk) + xStart;
                data->rz = (zc * zPerChunk) + zStart;
                data->rw = (xc * xPerChunk) + (xc == xTotalChunks ? xRangeRemainder : 0);
                data->rh = (zc * zPerChunk) + (zc == zTotalChunks ? zRangeRemainder : 0);

                cudaSafeCall(hipMalloc((void **)&dataDevice, sizeof(sfData)));
                cudaSafeCall(hipMemcpy(dataDevice, data, sizeof(sfData), hipMemcpyHostToDevice));

                deviceTask<<< numBlocks, threadsPerBlock >>>(dataDevice);
                
                cudaSafeCall(hipGetLastError());
                cudaSafeCall(hipDeviceSynchronize());

                cudaSafeCall(hipFree(dataDevice));
            }
        }
    }

    hipDeviceReset();
}